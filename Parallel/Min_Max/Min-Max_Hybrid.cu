#include "hip/hip_runtime.h"
/*
Need optimization add sorting
opt of 4-opt
bottlneck of sorting kernel, Modified 30-3-2019 heapSort added
*/
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include<thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include<hiprand/hiprand_kernel.h>
#include<thrust/extrema.h>
#include<thrust/device_ptr.h>
#include "Utility.cuh"

#define MAXBLOCKS 1<<30

using namespace std;

//Heap Sort
__global__ void heapsortBasedOnDegree(long int V,long int *preSum,long int *colIndex,long int *degree){
  
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(threadId<V){

        long int n = preSum[threadId+1]-preSum[threadId];
        long int largest,previous,temp;
        
        for (int i = preSum[threadId]+n/2-1;i>=preSum[threadId];i--){ 
            
            largest = i,previous=-1;     
            while(largest!=previous){
              int l = 2*largest-preSum[threadId]+1;
              int r = 2*largest-preSum[threadId]+2;
              previous = largest;
              if(l<preSum[threadId+1]&&degree[colIndex[l]]<degree[colIndex[largest]]) 
                  largest = l; 
     
              if(r<preSum[threadId+1]&&degree[colIndex[r]]<degree[colIndex[largest]]) 
                largest = r; 
     
              if(largest!=previous) 
              { 
                temp = colIndex[previous];
                colIndex[previous] = colIndex[largest];
                colIndex[largest] = temp;
              }
           }
     
        }
 
        // One by one extract an element from heap 
        for (int i=preSum[threadId+1]-1;i>=preSum[threadId];i--) 
        { 
            temp = colIndex[preSum[threadId]];
            colIndex[preSum[threadId]] = colIndex[i];
            colIndex[i] = temp;
            largest = preSum[threadId],previous=-1;
            while(largest!=previous){
              int l = 2*largest-preSum[threadId]+1;
              int r = 2*largest-preSum[threadId]+2;
              previous = largest;
              if(l<i&&degree[colIndex[l]]<degree[colIndex[largest]]) 
                  largest = l; 
     
              if(r<i&&degree[colIndex[r]]<degree[colIndex[largest]]) 
                largest = r; 
     
              if(largest!=previous) 
              { 
                temp = colIndex[previous];
                colIndex[previous] = colIndex[largest];
                colIndex[largest] = temp;
             
              }
           }
     
        } 
    }

}

//Heap Sort
__global__ void heapsortBasedOnRandom(long int V,long int *preSum,long int *colIndex,double *random){
  
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(threadId<V){

        long int n = preSum[threadId+1]-preSum[threadId];
        long int largest,previous,temp;
        
        for (int i = preSum[threadId]+n/2-1;i>=preSum[threadId];i--){ 
            
            largest = i,previous=-1;     
            while(largest!=previous){
              int l = 2*largest-preSum[threadId]+1;
              int r = 2*largest-preSum[threadId]+2;
              previous = largest;
              if(l<preSum[threadId+1]&&random[colIndex[l]]<random[colIndex[largest]]) 
                  largest = l; 
     
              if(r<preSum[threadId+1]&&random[colIndex[r]]<random[colIndex[largest]]) 
                largest = r; 
     
              if(largest!=previous) 
              { 
                temp = colIndex[previous];
                colIndex[previous] = colIndex[largest];
                colIndex[largest] = temp;
              }
           }
     
        }
 
        // One by one extract an element from heap 
        for (int i=preSum[threadId+1]-1;i>=preSum[threadId];i--) 
        { 
            temp = colIndex[preSum[threadId]];
            colIndex[preSum[threadId]] = colIndex[i];
            colIndex[i] = temp;
            largest = preSum[threadId],previous=-1;
            while(largest!=previous){
              int l = 2*largest-preSum[threadId]+1;
              int r = 2*largest-preSum[threadId]+2;
              previous = largest;
              if(l<i&&random[colIndex[l]]<random[colIndex[largest]]) 
                  largest = l; 
     
              if(r<i&&random[colIndex[r]]<random[colIndex[largest]]) 
                largest = r; 
     
              if(largest!=previous) 
              { 
                temp = colIndex[previous];
                colIndex[previous] = colIndex[largest];
                colIndex[largest] = temp;
             
              }
           }
     
        } 
    }

}


__global__ void generateRandom(hiprandState* globalState, double* randomArray,long int V,unsigned long seed) 
{
    long int idx = blockIdx.x*blockDim.x+threadIdx.x;
    
    if(idx<V){
       
       hiprandState localState = globalState[idx];
       hiprand_init(seed,idx,0,&localState);
       double RANDOM = hiprand_uniform(&localState);
       randomArray[idx] = RANDOM;
       globalState[idx] = localState;
    
    }

}


__global__ void assignColors(long int V,long int c,double *random,long int *colors,double *randomValue,
                             int *maxe,int *mine,int *left){
   
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;

    if(threadId<V&&colors[threadId]!=-1){
       
       return;
    
    }else if(threadId<V){
        
        if(colors[threadId]==-1){
            
            if(randomValue[threadId]==1){
                
                colors[threadId] = c;
                atomicAdd(maxe,1);
            
            }else if(randomValue[threadId]==-1){
                
                colors[threadId] = c+1;
                atomicAdd(mine,1);
            
            }else{
             
                *left = 1;
            
            }

        }

    }

}

__global__ void assign_DegreeColors(long int V,long int c,long int *degree,bool *flag,
                                    long int *colors,long int *degreeValue,bool *isDegree,int *maxe,int *mine,int *left){
    
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(threadId<V&&colors[threadId]!=-1){
       
       return;

    }else if(threadId<V){
        
        if(colors[threadId]==-1){
           
           if(flag[threadId]==false){
              
              if(degreeValue[threadId]==1){
                  
                  colors[threadId] = c;
                  *isDegree = true;
                  atomicAdd(maxe,1);
                  //reAssign Values
              
              }else if(degreeValue[threadId]==-1){
                  
                  colors[threadId] = c+1;
                  *isDegree = true;
                  atomicAdd(mine,1);
              
              }else{
                 
                  *left = 1;
              
              } 
           }else{
              
              flag[threadId] = false;
              *left = 1;
           
           }
        }

    }

}

//Degree Coloring
__global__ void degree_Coloring(long int V,long int c,long int *preSum,long int *colIndex,
                                long int *degree,long int *colors,long int *degreeValue,bool *d_flag){
  
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
   long int minmaxDegree;

   if(threadId<V&&colors[threadId]!=-1){
      
      return;

   }
   else if(threadId<V){
        
       if(colors[threadId]==-1){
          
           minmaxDegree = -1;
           long int start = preSum[threadId],end = preSum[threadId+1]-1;

           for(;start<=end;){
               long int j=colIndex[start];
               long int jc = colors[j];
               
               if(colors[j]!=-1){
                 start++;
                 continue;
               }

              //check all neighborhood which all are uncolored
              if(jc==-1){
                 if(degree[threadId]==degree[j]){
                    d_flag[threadId] = true;
                    break;  
                 }else if(degree[j]>minmaxDegree){
                   minmaxDegree = degree[j];
                   break;
                 }

                 start++;

              }

           }

           degreeValue[threadId] = degree[threadId]>minmaxDegree?1:0;
           
           if(degreeValue[threadId]==0&&d_flag[threadId]==false){
              
              minmaxDegree = LONG_MAX;
              for(;start<=end;){
                 long int j=colIndex[end];
                 long int jc = colors[j];
                 
                 if(colors[j]!=-1){
                   end--;
                   continue;
                 }

                //check all neighborhood which all are uncolored
                if(jc==-1){
                   if(degree[threadId]==degree[j]){
                      d_flag[threadId] = true;
                      break;  
                   }else if(degree[j]<minmaxDegree){
                      minmaxDegree = degree[j];
                      break;
                   }

                   end--;

                }

             }               
             
             degreeValue[threadId] = degree[threadId]<minmaxDegree?-1:0;
          }

       }  
        
   }

}

// How to arrange the program to assign the different color to adjacent
__global__ void minmax_Based_Label(long int V,long int c,long int *preSum,long int *colIndex,
                                   double *random,long int *colors,double *randomValue){
     
     long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
     double minMaxValue;
     
     if(threadId<V&&colors[threadId]!=-1){
        return;
     }
     else if(threadId<V){
              
         if(colors[threadId]==-1){
          
            minMaxValue = -1;
            long int start = preSum[threadId],end = preSum[threadId+1]-1,jcs,jce,js,je;
            
            for(;start<=end;){

                js = colIndex[start];
                jcs = colors[js];
                
                if(jcs!=-1){
                  start++;
                  continue;
                }
                else if(jcs==-1){       
                  //check with random value
                   if(random[js]>minMaxValue){
                      minMaxValue = random[js];
                      break;
                   }
                   start++;
                }
             
            }
            
            randomValue[threadId] = random[threadId]>minMaxValue?1:0;
            
            if(randomValue[threadId]==0){
              
              minMaxValue = LONG_MAX;

              for(;start<=end;){

                  je = colIndex[end];
                  jce = colors[je];
                  
                  if(jce!=-1){
                    end--;
                    continue;
                  }else if(jce==-1){

                    //check with random value
                    if(random[je]<minMaxValue){
                      minMaxValue = random[je];
                      break;
                    }

                    end--;

                   }
               
              }
              
              randomValue[threadId] = random[threadId]<minMaxValue?-1:0;
            
            }

     
          }

     }
} 

int preSumLength(int V,long int *d_preSum,long int *degree){
    
    long int prev = degree[0];
    int flag = 0;

    for(long int i=0;i<V;i++){
       d_preSum[i+1] = d_preSum[i]+degree[i];
       if(prev!=degree[i]){
         flag = 1;
       }
    }
    return flag;
}

__global__ void IsValidgraph_Coloring(long int V,long int *colors,long int *preSum,long int *colIndex,bool *flag){
   
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
   if(threadId<V){
       for(long int i=preSum[threadId];i<preSum[threadId+1];i++){
          if(colors[threadId]==colors[colIndex[i]]||colors[threadId]==-1){
             *flag = false;
          }
       }
   }
   
}


long int Min_Max_Algorithm(long int V,long int *preSum,long int *colIndex,long int *colors,
                           long int *degree,int isSwitching){
    hiprandState* devStates;
    double *d_random;
    hipMallocManaged(&d_random,sizeof(double)*V);
    long int *d_degreeValue;
    double *d_randomValue;
    bool *d_flag;
    bool *d_isDegree,isDegree;
    int *d_mine,*d_maxe,*d_left,left,maxe,mine;

    //Allocate the memory for maxArray
    catchCudaError(hipMallocManaged(&d_randomValue,sizeof(double)*V),"d_randomValue Allocation");
    catchCudaError(hipMallocManaged(&d_degreeValue,sizeof(long int)*V),"d_degreeValue Allocation");
    catchCudaError(hipMallocManaged(&d_flag,sizeof(bool)*V),"Flag Allocation");
    catchCudaError(hipMallocManaged(&d_isDegree,sizeof(bool)),"Check Allocation");
    catchCudaError(hipMallocManaged(&d_maxe,sizeof(int)),"maxe Allocation");
    catchCudaError(hipMallocManaged(&d_mine,sizeof(int)),"mine Allocation");
    catchCudaError(hipMallocManaged(&d_left,sizeof(int)),"left Allocation");
    hipMallocManaged(&devStates,V*sizeof(hiprandState));

    long int n_threads =  256;
    long int n_blocks = min((V+n_threads-1)/n_threads,(long)MAXBLOCKS);
    /*
    @ step 2 Initialize the colors to -1
    @ until all are colored
    */
    thrust::fill(colors,colors+V,-1);
    long int cr = 1;

    //Default case;
    isDegree = false;
    clock_t gpu_time = clock();

    //Step 1 assign the random value to all the vertices
    if(isSwitching){
       generateRandom<<<n_blocks,n_threads>>>(devStates,d_random,V,time(NULL));
       heapsortBasedOnDegree<<<n_blocks,n_threads>>>(V,preSum,colIndex,degree);
    }else{
       generateRandom<<<n_blocks,n_threads>>>(devStates,d_random,V,time(NULL));
       heapsortBasedOnRandom<<<n_blocks,n_threads>>>(V,preSum,colIndex,d_random);
    }
    
    do{
      
       //For Degree Coloring
       if(isDegree||isSwitching){
          
          isDegree = false;
          mine = 0;
          maxe = 0;
          left = 0;
          catchCudaError(hipMemcpy(d_isDegree,&isDegree,sizeof(bool),hipMemcpyHostToDevice),"isDegreeD Copy");
          catchCudaError(hipMemcpy(d_maxe,&maxe,sizeof(int),hipMemcpyHostToDevice),"maxeD copy");
          catchCudaError(hipMemcpy(d_maxe,&mine,sizeof(int),hipMemcpyHostToDevice),"mineD copy");
          catchCudaError(hipMemcpy(d_left,&left,sizeof(int),hipMemcpyHostToDevice),"leftD copy");
          degree_Coloring<<<n_blocks,n_threads>>>(V,cr,preSum,colIndex,degree,colors,d_degreeValue,d_flag);       
          assign_DegreeColors<<<n_blocks,n_threads>>>(V,cr,degree,d_flag,colors,d_degreeValue,d_isDegree,d_maxe,d_mine,d_left);
          catchCudaError(hipMemcpy(&isDegree,d_isDegree,sizeof(bool),hipMemcpyDeviceToHost),"isDegree Copy");
          catchCudaError(hipMemcpy(&maxe,d_maxe,sizeof(int),hipMemcpyDeviceToHost),"maxe copy");
          catchCudaError(hipMemcpy(&mine,d_mine,sizeof(int),hipMemcpyDeviceToHost),"mine copy");;
          catchCudaError(hipMemcpy(&left,d_left,sizeof(int),hipMemcpyDeviceToHost),"left copy");
          
          cr = (maxe&&mine)?cr+2:(maxe||mine)?cr+1:cr;

          if(isDegree==false){
            isSwitching = 0;
            heapsortBasedOnRandom<<<n_blocks,n_threads>>>(V,preSum,colIndex,d_random);
          }

       }else{
         
         //For Randomized Based Labelling
         maxe = 0;
         mine = 0;
         left = 0;
         catchCudaError(hipMemcpy(d_left,&left,sizeof(int),hipMemcpyHostToDevice),"left copy");
         catchCudaError(hipMemcpy(d_maxe,&maxe,sizeof(int),hipMemcpyHostToDevice),"maxe copy");
         catchCudaError(hipMemcpy(d_maxe,&mine,sizeof(int),hipMemcpyHostToDevice),"mine copy");
         minmax_Based_Label<<<n_blocks,n_threads>>>(V,cr,preSum,colIndex,d_random,colors,d_randomValue);       
         assignColors<<<n_blocks,n_threads>>>(V,cr,d_random,colors,d_randomValue,d_maxe,d_mine,d_left);
         catchCudaError(hipMemcpy(&left,d_left,sizeof(int),hipMemcpyDeviceToHost),"left copy");
         catchCudaError(hipMemcpy(&maxe,d_maxe,sizeof(int),hipMemcpyDeviceToHost),"maxe copy");
         catchCudaError(hipMemcpy(&mine,d_mine,sizeof(int),hipMemcpyDeviceToHost),"mine copy");
         
         cr = (maxe&&mine)?cr+2:(maxe||mine)?cr+1:cr;

       }             
    
    }while(left);
    gpu_time = clock()-gpu_time;
    //Assigned Colors
    /*
    @ last step to print the assigned colors
    */
    printf("\n");
    for(long int i=0;i<V;i++){
       printf("vertex --> %i Assigned Color --> %d\n",i,colors[i]);
    }
    printf("\n");
    printf("gpu_time is %.6lf\n",(double)gpu_time/((double)CLOCKS_PER_SEC/1000));

    //thrust::device_ptr<long int> d_ptr = thrust::device_pointer_cast(colors);
    //long int minimumColor = *(thrust::max_element(d_ptr, d_ptr+V));
    thrust::device_vector<long int> d_data(V);
    thrust::copy(colors,colors+V,d_data.begin());
    thrust::sort(d_data.begin(), d_data.end());

    size_t num_unique = thrust::inner_product(d_data.begin(), d_data.end()-1,d_data.begin()+1,0,
                                              thrust::plus<long int>(),thrust::not_equal_to<long int>())+1;

    hipFree(d_random);
    hipFree(d_flag);
    hipFree(d_randomValue);
    hipFree(d_degreeValue);
    hipFree(d_isDegree);
    hipFree(d_left);
    hipFree(d_maxe);
    hipFree(d_mine);
    
    //required colors needed
    return (long int)num_unique;
}

void GraphColoring_GPUAllocation(const char filename[]){
   
   //@difficult to allocate memory for large complete dataset not assume complete graph
   long int V; //No. of verties
   long int n_zero_counter = 0;   
   long int **st_Column;
   long int *st_degree;
   
   if(string(filename).find("col")!=string::npos||string(filename).find("clq")!=string::npos){
     ReadColFile(filename,&V,&st_Column,&st_degree,&n_zero_counter);
   }else{
     ReadMMFile(filename,&V,&st_Column,&st_degree,&n_zero_counter); 
   }
   
   long int *degree;
   catchCudaError(hipMallocManaged(&degree,sizeof(long int)*V),"Degree Allocation");
   thrust::copy(st_degree,st_degree+V,degree);
   
   long int *d_preSum;
   catchCudaError(hipMallocManaged(&d_preSum,sizeof(long int)*(V+1)),"preSum Allocation");
   d_preSum[0] = 0;
   //store all the index of non zero element
   long int *d_colIndex;
   catchCudaError(hipMallocManaged(&d_colIndex,sizeof(long int)*n_zero_counter),"colIndex Allocation");
   
   //Allocation
   long int *colors;
   catchCudaError(hipMallocManaged(&colors,sizeof(long int)*V),"Color Allocation");

   int isSwitching = preSumLength(V,d_preSum,degree);

   for(int i=0;i<V;i++){
      //Remove the hipMemcpy it will take more time
      thrust::copy(st_Column[i],st_Column[i]+degree[i],d_colIndex+d_preSum[i]);
   }  
   
   /*
   @begin CSR
   */
   long int width=16,height=16;
   long int threads_per_blocks = width*height;
   
   //Call the Min-Max Algorithm
   long int number_Of_Colors_Needed = Min_Max_Algorithm(V,d_preSum,d_colIndex,colors,degree,isSwitching);  
   
   printf("Min-Max Algorithm coloring found solution with %d colors\n",number_Of_Colors_Needed);
   printf("Valid coloring "); 
   
   bool *d_isValidColors; 
   catchCudaError(hipMallocManaged(&d_isValidColors,sizeof(bool)),"IsValid Allocation");
   *d_isValidColors = true;
   
   IsValidgraph_Coloring<<<ceil(V/threads_per_blocks)+1,threads_per_blocks>>>(V,colors,d_preSum,d_colIndex,d_isValidColors); 
   catchCudaError(hipMemcpy(d_isValidColors,d_isValidColors,sizeof(bool),hipMemcpyDeviceToHost),"IsValid Host Copy");
   
   if(*d_isValidColors){
     printf("yes\n");
   }else{
     printf("No");
   }
   
   //catchCudaError(hipDeviceSynchronize(),"GraphColoring DeviceSync");
   
   hipFree(d_preSum);
   hipFree(d_colIndex);
   hipFree(colors);
   hipFree(degree);
   //hipFree(d_isValidColors);
  
}

/* Reading Argument with command line opetion */
int main(int argc,char *argv[])
{
     if(argc<2){
       printf("Invalid Input Parameter\n");
       exit(1);
     }else{
      
     /*
     @Adding the clock
     */
     clock_t time = clock();
     GraphColoring_GPUAllocation(argv[1]);
     time = clock()-time; 
     
     double execution_time = (double)time/(double)CLOCKS_PER_SEC;
     
     printf("Total execution time is %lf\n",execution_time);
    
   }   

   return 0;
}
