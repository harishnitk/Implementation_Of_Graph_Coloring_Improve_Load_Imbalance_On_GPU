#include "hip/hip_runtime.h"
/*
# Edge Base Approach
#{class} {use the bit manipulations for less memeory requirements}
@iterative
*/

#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include<thrust/count.h>
#include<thrust/extrema.h>
#include<thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include<hiprand/hiprand_kernel.h>
#include "Utility.cuh"

#define MAXBLOCKS 1<<32
#define MOD 32

using namespace std;

__global__ void AssignColors(long int V,long int *preSum,long int *colIndex,long int *colors,long int *delta_Degree,bool *conflicts){
  
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
   long int stride = blockDim.x*gridDim.x;
   
   if(threadId<V&&!conflicts[threadId]){
      return;
   }


   if(threadId<V){
      
     for(long int i=threadId;i<V;i=i+stride){

        long int *vforbidden = (long int*)malloc(sizeof(long int)*(*delta_Degree+1));
        memset(vforbidden,0,sizeof(long int)*(*delta_Degree+1));

        for(long int k=preSum[i];k<preSum[i+1];k++){

           long int j = colIndex[k];
           long int value = colors[j]%MOD;
           long int shift = 1<<value;
           vforbidden[colors[j]/MOD]|= shift; 
        }
        
        //Assign colors
        for(long int color=1;color<=*delta_Degree+1;color++){

            long int val = color%MOD;
            
            if((vforbidden[color/MOD]&(1<<val))== 0){
                colors[i] = color;
                return;
            }

        }

       
        free(vforbidden);
        
     }

   }

}

__global__ void DetectConflicts(long int V,long int *preSum,long int *colIndex,long int *colors,bool *conflicts,bool *checkConflicts){

    long int threadId = blockIdx.x*blockDim.x+threadIdx.x;

    if(threadId<V){

        conflicts[threadId] = false;

        for(long int k=preSum[threadId];k<preSum[threadId+1];k++){

            long int j = colIndex[k];

            if((colors[threadId]==colors[j])&&(j<threadId)){

                conflicts[threadId] = true;
                *checkConflicts = true;
                return;

            }
        }
    
    }
  
}

__global__ void preSumLength(int V,long int *d_preSum,long int *degree,long int *delta_Degree){
   
    for(long int i=0;i<V;i++){
       
       d_preSum[i+1] = d_preSum[i]+degree[i];
       
       if(*delta_Degree<degree[i]){
          *delta_Degree = degree[i];
       }
    }
}

__global__ void IsValidgraph_Coloring(long int V,long int *colors,long int *preSum,long int *colIndex,bool *flag){
   
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
   if(threadId<V){
       for(long int i=preSum[threadId];i<preSum[threadId+1];i++){
          if(colors[threadId]==colors[colIndex[i]]||colors[threadId]==-1){
             *flag = false;
          }
       }
   }
   
}


long int EdgeBased_Algorithm(long int V,long int *preSum,long int *colIndex,long int *colors,long int *degree,long int n_zero_counter,long int *delta_Degree){
    
    /*
    @ step 2 Initialize the colors to 0
    @ until all are colored
    */
    thrust::fill(colors,colors+V,0);
    
    long int n_threads =  256;
    long int n_blocks = min((V+n_threads-1)/n_threads,(long)MAXBLOCKS);
    bool *d_conflicts,*checkConflict;
    hipMallocManaged(&d_conflicts,sizeof(bool)*V);
    hipMallocManaged(&checkConflict,sizeof(bool));
    thrust::fill(d_conflicts,d_conflicts+V,true);
    
    do{
       
       *checkConflict = false;

       AssignColors<<<n_blocks,n_threads>>>(V,preSum,colIndex,colors,delta_Degree,d_conflicts);
       DetectConflicts<<<n_blocks,n_threads>>>(V,preSum,colIndex,colors,d_conflicts,checkConflict);
       
       catchCudaError(hipDeviceSynchronize(),"Edge");
        

    }while(*checkConflict);
    
    //Assigned Colors
    /*
    @ last step to print the assigned colors
    */
    cout<<endl;
    for(long int i=0;i<V;i++){
       printf("vertex --> %i Assigned Color --> %d\n",i,colors[i]);
    }
    cout<<endl;

    //thrust::device_ptr<long int> d_ptr = thrust::device_pointer_cast(colors);
    //long int minimumColor = *(thrust::max_element(d_ptr, d_ptr+V));
    thrust::device_vector<long int> d_data(V);
    thrust::copy(colors,colors+V,d_data.begin());
    thrust::sort(d_data.begin(), d_data.end());

    size_t num_unique = thrust::inner_product(d_data.begin(), d_data.end()-1,d_data.begin()+1,0,
                                              thrust::plus<long int>(),thrust::not_equal_to<long int>())+1;

    hipFree(d_conflicts);
    hipFree(checkConflict);
    
    //required colors needed
    return (long int)num_unique;
}

void GraphColoring_GPUAllocation(const char filename[]){
   
   //@difficult to allocate memory for large complete dataset not assume complete graph
   long int V; //No. of verties
   long int n_zero_counter = 0;   
   long int **st_Column;
   long int *st_degree;
   
   if(string(filename).find("col")!=string::npos){
     ReadColFile(filename,&V,&st_Column,&st_degree,&n_zero_counter);
   }else{
     ReadMMFile(filename,&V,&st_Column,&st_degree,&n_zero_counter); 
   }
   
   long int *degree;
   catchCudaError(hipMallocManaged(&degree,sizeof(long int)*V),"Degree Allocation");
   thrust::copy(st_degree,st_degree+V,degree);
   
   long int *d_preSum;
   catchCudaError(hipMallocManaged(&d_preSum,sizeof(long int)*(V+1)),"preSum Allocation");
   d_preSum[0] = 0;
   //store all the index of non zero element
   long int *d_colIndex;
   catchCudaError(hipMallocManaged(&d_colIndex,sizeof(long int)*n_zero_counter),"colIndex Allocation");
   
   //Allocation
   long int *colors;
   catchCudaError(hipMallocManaged(&colors,sizeof(long int)*V),"Color Allocation");
   long int *delta_Degree;
   catchCudaError(hipMallocManaged(&delta_Degree,sizeof(long int)),"Delta Degree Allocation");
   *delta_Degree = 0;

   preSumLength<<<1,1>>>(V,d_preSum,degree,delta_Degree);
   catchCudaError(hipMemcpy(d_preSum,d_preSum,sizeof(long int)*(V+1),hipMemcpyDeviceToHost),"Copy to PreSum");

   for(int i=0;i<V;i++){
      //Remove the hipMemcpy it will take more time
      thrust::copy(st_Column[i],st_Column[i]+degree[i],d_colIndex+d_preSum[i]);
   }  
   
   /*
   @begin CSR
   */
   
   //Call the EdgeBase Algorithm
   long int number_Of_Colors_Needed = EdgeBased_Algorithm(V,d_preSum,d_colIndex,colors,degree,n_zero_counter,delta_Degree);  
   
   cout<<"EdgeBase Algorithm coloring found solution with "<<number_Of_Colors_Needed<<" colors"<<endl;
   cout<<"Valid coloring Yes\n"; 
   
   catchCudaError(hipDeviceSynchronize(),"GraphColoring DeviceSync");
   

   hipFree(d_preSum);
   hipFree(d_colIndex);
   hipFree(colors);
   hipFree(degree);
}

/* Reading Argument with command line opetion */
int main(int argc,char *argv[])
{
     if(argc<2){
       cout<<"Invalid Input Parameter"<<endl;
       exit(1);
     }else{
      
     /*
     @Adding the clock
     */
     clock_t time = clock();
     GraphColoring_GPUAllocation(argv[1]);
     time = clock()-time; 
     
     cout<<"Total execution time is "<<(double)time/(double)CLOCKS_PER_SEC<<endl;
    
   }   

   return 0;
}
