#include "hip/hip_runtime.h"
/*
# Edge Base Approach
#{class} {use the bit manipulations for less memeory requirements}
@working Latest more fast but give more color than previous
@Recolor function Added 21-03-2019, resolve Degree Based
*/

#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include<thrust/count.h>
#include<thrust/extrema.h>
#include<thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include<hiprand/hiprand_kernel.h>
#include "Utility.cuh"

#define MAXBLOCKS 1<<32
#define MOD 32

using namespace std;


__global__ void AssignColors(long int V,long int *delta_Degree,long int *colIndex,long int *colors,
                             unsigned int *CS,unsigned long long int *vforbidden){
  
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;

   if(threadId<V){
      
      //conflicts colors
      if(colors[threadId]==0){
        
        //First Available Color
        if(vforbidden[threadId]==0){

           colors[threadId] = CS[threadId]+1;
        
        }else{
           CS[threadId] = CS[threadId]+1;
           vforbidden[threadId] = 0;
        }

      }

   }


}

__global__ void DetectConflicts(long int V,long int *preSum,long int *colIndex,long int *colors,
                                bool *checkConflict,long int *degree,long int *modify){

    long int threadId = blockIdx.x*blockDim.x+threadIdx.x;

    if(threadId<V){
        
        modify[threadId] = 0;

        if(colors[threadId]==0){
           *checkConflict = true;
           return;
        }

        for(long int k=preSum[threadId];k<preSum[threadId+1];k++){

            long int j = colIndex[k];

            if(colors[j]==0){
               continue;
            }

            if((colors[threadId]==colors[j])&&degree[j]<degree[threadId]){

                colors[j] = 0;
                *checkConflict = true;

            }else if((colors[threadId]==colors[j])&&degree[j]>degree[threadId]){

                colors[threadId] = 0;
                *checkConflict = true;
                return;

            }else if((colors[threadId]==colors[j])&&j<threadId){

                colors[j] = 0;
                *checkConflict = true;

            }else if((colors[threadId]==colors[j])&&j>threadId){

                colors[threadId] = 0;
                *checkConflict = true;
                return;

            }
        }
    
    }
  
}

__global__ void ForbiddenColors(long int V,long int *preSum,long int *colIndex,long int *colors,
                                unsigned long long int *vforbidden,unsigned int *CS,int flag){
 
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;

   if(threadId<V){
      
      if(colors[threadId]){
         return;
      }  
      
      long int maxColor = LONG_MIN;
      
      for(long int k=preSum[threadId];k<preSum[threadId+1];k++){
          
          long int j = colIndex[k];
          
          if(colors[j]==0){
            continue;
          }
          
          if(CS[j]==CS[threadId]){
             
             if(colors[j]!=0){
                unsigned long long int value = (vforbidden[threadId]|colors[j])-vforbidden[threadId];
                atomicAdd(&vforbidden[threadId],value);
             }
          }

          if(colors[j]!=0&&maxColor<colors[j]){
             maxColor = colors[j];
          }        

      }
      
      if(flag==0){
        if((colors[threadId]==0)){//&&(vforbidden[threadId]==0)){
           CS[threadId] = maxColor;
        }
      }
   
   }
  
}

/*
@Larger degree first approch
*/
__global__ void resolveAdj_LargeIndex(long int V,long int *preSum,long int *colIndex,long int *colors,
                              unsigned long long int *vforbidden,unsigned int *CS,bool *change,int flag,long int *degree,long int *modify){

     long int node = blockDim.x*blockIdx.x+threadIdx.x;

     if(node<V){

        if(vforbidden[node]==0||colors[node]||modify[node]==1){
           return;
        }
        
        if(flag){
          
          long int larger=node,seclarger=node,start=preSum[node],end=preSum[node+1]-1,maxColor=LONG_MIN;

          for(;start<=end;){
              
              long int j = colIndex[start];
              
              if((CS[node]==CS[j])&&colors[j]==0&&degree[j]>degree[node]&&modify[j]==0){
                 
                 if(larger<j&&degree[larger]<degree[j]){
                    seclarger = larger;
                    larger = j;
                    *change = true;
                    CS[larger] = CS[larger]+1;
                    if(modify[seclarger]==0){
                      CS[seclarger] = CS[seclarger]+1;  
                    }
                    modify[seclarger] = 1;
                    modify[larger] = 1;
                 }else if(degree[seclarger]<degree[j]){
                    seclarger = j;
                    modify[seclarger] = 1;
                    CS[seclarger] = CS[seclarger]+1;
                 }
                 

              }else if(colors[j]!=0&&maxColor<colors[j]){
                 maxColor = colors[j];
              }

              j = colIndex[end];
              if((CS[node]==CS[j])&&colors[j]==0&&degree[j]>degree[node]&&modify[j]==0){
                 
                 if(larger<j&&degree[larger]<degree[j]){
                    seclarger = larger;
                    larger = j;
                    *change = true;
                    CS[larger] = CS[larger]+1;
                    if(modify[seclarger]==0){
                      CS[seclarger] = CS[seclarger]+1;  
                    }
                    modify[seclarger] = 1;
                    modify[larger] = 1;
                 }else if(degree[seclarger]<degree[j]){
                    seclarger = j;
                    modify[seclarger] = 1;
                    CS[seclarger] = CS[seclarger]+1;
                 }
                 

              }else if(colors[j]!=0&&maxColor<colors[j]){
                 maxColor = colors[j];
              }

              start++;
              end--;

          }

          if(larger!=seclarger){
             colors[larger] = maxColor+1;
             colors[seclarger] = colors[larger]+1;
          }else if(modify[node]==0){
             modify[node] = 1;
             CS[node] = CS[node]+1;
          }

        }

     }

}

void preSumLength(int V,long int *d_preSum,long int *degree,long int *delta_Degree){
   
    for(long int i=0;i<V;i++){
       
       d_preSum[i+1] = d_preSum[i]+degree[i];
       
       if(*delta_Degree<degree[i]){
          *delta_Degree = degree[i];
       }
    }

    *delta_Degree = *delta_Degree+1;
}

__global__ void IsValidgraph_Coloring(long int V,long int *colors,long int *preSum,long int *colIndex,bool *flag){
   
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
   if(threadId<V){
       for(long int i=preSum[threadId];i<preSum[threadId+1];i++){
          if(colors[threadId]==colors[colIndex[i]]||colors[threadId]==-1){
             *flag = false;
          }
       }
   }
   
}

long int EdgeBased_Algorithm(long int V,long int *preSum,long int *colIndex,long int *colors,long int *degree,long int n_zero_counter,long int *delta_Degree){
    
    unsigned int *CS;
    hipMallocManaged(&CS,sizeof(unsigned int)*V);
    unsigned long long int *d_vforbidden;
    hipMallocManaged(&d_vforbidden,sizeof(unsigned long long int)*V);
    thrust::fill(d_vforbidden,d_vforbidden+V,0); 
    thrust::fill(CS,CS+V,0);
    bool *d_change;
    int flag=1;
    hipMallocManaged(&d_change,sizeof(bool));
    long int *d_modify;
    hipMallocManaged(&d_modify,sizeof(long int)*V);
    
    /*
    @ step 2 Initialize the colors to 0
    @ until all are colored
    */
    thrust::fill(colors,colors+V,0);
    
    long int n_threads =  256;
    long int n_blocks = min((V+n_threads-1)/n_threads,(long)MAXBLOCKS);
    bool *checkConflict;
    hipMallocManaged(&checkConflict,sizeof(bool));  
        
    clock_t gpu_time = clock();
    do{
       
       *checkConflict = false;
       *d_change = false;

       AssignColors<<<n_blocks,n_threads>>>(V,delta_Degree,colIndex,colors,CS,d_vforbidden);
       DetectConflicts<<<n_blocks,n_threads>>>(V,preSum,colIndex,colors,checkConflict,degree,d_modify);
       ForbiddenColors<<<n_blocks,n_threads>>>(V,preSum,colIndex,colors,d_vforbidden,CS,flag);
       catchCudaError(hipMemcpy(checkConflict,checkConflict,sizeof(bool),hipMemcpyDeviceToHost),"Conflict Copy");
  
       if(flag){
          
          resolveAdj_LargeIndex<<<n_blocks,n_threads>>>(V,preSum,colIndex,colors,d_vforbidden,CS,d_change,flag,degree,d_modify);
          catchCudaError(hipMemcpy(d_change,d_change,sizeof(bool),hipMemcpyDeviceToHost),"d_change Copy");
          
          if(*d_change==false){
             flag = 0;
          }

       }


    }while(*checkConflict);
    gpu_time = clock()-gpu_time;
    //Assigned Colors
    /*
    @ last step to print the assigned colors
    */
    
    cout<<endl;
    for(long int i=0;i<V;i++){
       printf("vertex --> %i Assigned Color --> %d\n",i,colors[i]);
    }
    cout<<endl;
    printf("gpu_time is %.6lf\n",(double)gpu_time/((double)CLOCKS_PER_SEC/1000));   

    //thrust::device_ptr<long int> d_ptr = thrust::device_pointer_cast(colors);
    //long int minimumColor = *(thrust::max_element(d_ptr, d_ptr+V));
    thrust::device_vector<long int> d_data(V);
    thrust::copy(colors,colors+V,d_data.begin());
    thrust::sort(d_data.begin(), d_data.end());

    size_t num_unique = thrust::inner_product(d_data.begin(), d_data.end()-1,d_data.begin()+1,0,
                                              thrust::plus<long int>(),thrust::not_equal_to<long int>())+1;

    hipFree(checkConflict);
    hipFree(d_vforbidden);
    hipFree(CS);

    //required colors needed
    return (long int)num_unique;//minimumColor;
}

void GraphColoring_GPUAllocation(const char filename[]){
   
   //@difficult to allocate memory for large complete dataset not assume complete graph
   long int V; //No. of verties
   long int n_zero_counter = 0;   
   long int **st_Column;
   long int *st_degree;
   
   if(string(filename).find("col")!=string::npos){
     ReadColFile(filename,&V,&st_Column,&st_degree,&n_zero_counter);
   }else{
     ReadMMFile(filename,&V,&st_Column,&st_degree,&n_zero_counter); 
   }
   
   long int *degree;
   catchCudaError(hipMallocManaged(&degree,sizeof(long int)*V),"Degree Allocation");
   thrust::copy(st_degree,st_degree+V,degree);
   
   long int *d_preSum;
   catchCudaError(hipMallocManaged(&d_preSum,sizeof(long int)*(V+1)),"preSum Allocation");
   d_preSum[0] = 0;
   //store all the index of non zero element
   long int *d_colIndex;
   catchCudaError(hipMallocManaged(&d_colIndex,sizeof(long int)*n_zero_counter),"colIndex Allocation");

   //Allocation
   long int *colors;
   catchCudaError(hipMallocManaged(&colors,sizeof(long int)*V),"Color Allocation");
   long int *delta_Degree;
   catchCudaError(hipMallocManaged(&delta_Degree,sizeof(long int)),"Delta Degree Allocation");
   *delta_Degree = 0;

   preSumLength(V,d_preSum,degree,delta_Degree);
   
   for(int i=0;i<V;i++){
      //Remove the hipMemcpy it will take more time
      thrust::copy(st_Column[i],st_Column[i]+degree[i],d_colIndex+d_preSum[i]);
   }  
   
   /*
   @begin CSR
   */
   long int width=16,height=16;
   long int threads_per_blocks = width*height;
   clock_t t = clock();
   //Call the EdgeBase Algorithm
   long int number_Of_Colors_Needed = EdgeBased_Algorithm(V,d_preSum,d_colIndex,colors,degree,n_zero_counter,delta_Degree);  
   
   cout<<"EdgeBase Algorithm coloring found solution with "<<number_Of_Colors_Needed<<" colors"<<endl;
   cout<<"Valid coloring "; 
   
   bool *d_isValidColors; 
   catchCudaError(hipMallocManaged(&d_isValidColors,sizeof(bool)),"IsValid Allocation");
   *d_isValidColors = true;

   IsValidgraph_Coloring<<<ceil(V/threads_per_blocks)+1,threads_per_blocks>>>(V,colors,d_preSum,d_colIndex,d_isValidColors); 
   catchCudaError(hipMemcpy(d_isValidColors,d_isValidColors,sizeof(bool),hipMemcpyDeviceToHost),"IsValid Host Copy");
   
   if(*d_isValidColors){
     cout<<"yes"<<endl;
   }else{
     cout<<"No"<<endl;
   }
   
   //catchCudaError(hipDeviceSynchronize(),"GraphColoring DeviceSync");

   hipFree(d_preSum);
   hipFree(d_colIndex);
   hipFree(colors);
   hipFree(degree);
   hipFree(d_isValidColors);
  
}

/* Reading Argument with command line opetion */
int main(int argc,char *argv[])
{
     if(argc<2){
       cout<<"Invalid Input Parameter"<<endl;
       exit(1);
     }else{
      
     /*
     @Adding the clock
     */
     clock_t time = clock();
     GraphColoring_GPUAllocation(argv[1]);
     time = clock()-time; 
     
     cout<<"Total execution time is "<<(double)time/(double)CLOCKS_PER_SEC<<endl;
    
   }   

   return 0;
}
