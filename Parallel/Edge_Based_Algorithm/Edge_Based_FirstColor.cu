#include "hip/hip_runtime.h"
/*
# Edge Base Approach
#{class} {use the bit manipulations for less memeory requirements}
@Working forbidden modified if condition revome cudaSynchronize in function
@Algo 4-opt Modified on 20-03-2019
@Add Degree Base
*/

#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include<thrust/count.h>
#include<thrust/extrema.h>
#include<thrust/device_ptr.h>
#include<hiprand/hiprand_kernel.h>
#include "Utility.cuh"

#define MAXBLOCKS 1<<32
#define MOD 32

using namespace std;

__global__ void AssignColors(long int V,long int *delta_Degree,long int *colIndex,long int *colors,
                             long int *CS,unsigned long long int *vforbidden){
  
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;

   if(threadId<V){
      
      //conflicts colors
      if(colors[threadId]==0){
        
        //First Available Color
        if(vforbidden[threadId]==0){

           colors[threadId] = CS[threadId]+1;
        
        }else{
           CS[threadId] = CS[threadId]+1;
           vforbidden[threadId] = 0;
        }

      }

   }


}

__global__ void DetectConflicts(long int V,long int *preSum,long int *colIndex,long int *colors,
                bool *checkConflict,unsigned long long int *vforbidden,long int *deltaDegree,long int *degree){

    long int threadId = blockIdx.x*blockDim.x+threadIdx.x;

    if(threadId<V){

        if(colors[threadId]==0){
           *checkConflict = true;
        }

        for(long int k=preSum[threadId];k<preSum[threadId+1];k++){

            long int j = colIndex[k];

            if((colors[threadId]!=0)&&(colors[j]!=0)&&(colors[threadId]==colors[j])&&(degree[j]>degree[threadId])){

                colors[threadId] = 0;
                *checkConflict = true;
                return;

            }else if((colors[threadId]!=0)&&(colors[j]!=0)&&(colors[threadId]==colors[j])&&(j>threadId)){

                colors[threadId] = 0;
                *checkConflict = true;
                return;

            }
        }
    
    }
  
}

__global__ void ForbiddenColors(long int V,long int *preSum,long int *colIndex,long int *colors,
                                unsigned long long int *vforbidden,long int *CS){
 
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;

   if(threadId<V&&colors[threadId]==0){
      
      for(long int k=preSum[threadId];k<preSum[threadId+1];k++){
          
          long int j = colIndex[k];

          if(CS[j]==CS[threadId]){
             
             if(colors[j]!=0&&colors[threadId]==0){
                unsigned long long int value = (vforbidden[threadId]|colors[j])-vforbidden[threadId];
                atomicAdd(&vforbidden[threadId],value);
             }else if(colors[j]==0&&colors[threadId]!=0){
                unsigned long long int value = (vforbidden[j]|colors[threadId])-vforbidden[j];
                atomicAdd(&vforbidden[j],value);
             }
          }        

      }
   
   }
  
}

void preSumLength(int V,long int *d_preSum,long int *degree,long int *delta_Degree){
   
    for(long int i=0;i<V;i++){
       
       d_preSum[i+1] = d_preSum[i]+degree[i];
       
       if(*delta_Degree<degree[i]){
          *delta_Degree = degree[i];
       }
    }

    *delta_Degree = *delta_Degree+1;
}

__global__ void IsValidgraph_Coloring(long int V,long int *colors,long int *preSum,long int *colIndex,bool *flag){
   
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
   if(threadId<V){
       for(long int i=preSum[threadId];i<preSum[threadId+1];i++){
          if(colors[threadId]==colors[colIndex[i]]||colors[threadId]==-1){
             *flag = false;
          }
       }
   }
   
}


long int EdgeBased_Algorithm(long int V,long int *preSum,long int *colIndex,long int *colors,long int *degree,long int n_zero_counter,long int *delta_Degree){
    
    long int *CS;
    hipMallocManaged(&CS,sizeof(long int)*V);
    unsigned long long int *d_vforbidden;
    hipMallocManaged(&d_vforbidden,sizeof(unsigned long long int)*V);
    thrust::fill(d_vforbidden,d_vforbidden+V,0); 
    thrust::fill(CS,CS+V,0);

    /*
    @ step 2 Initialize the colors to 0
    @ until all are colored
    */
    thrust::fill(colors,colors+V,0);
    
    long int minimumColor;
    long int n_threads =  256;
    long int n_blocks = min((V+n_threads-1)/n_threads,(long)MAXBLOCKS);
    bool *checkConflict;
    hipMallocManaged(&checkConflict,sizeof(bool));
    clock_t gpu_time = clock();

    do{
       
       *checkConflict = false;

       AssignColors<<<n_blocks,n_threads>>>(V,delta_Degree,colIndex,colors,CS,d_vforbidden);
       DetectConflicts<<<n_blocks,n_threads>>>(V,preSum,colIndex,colors,checkConflict,d_vforbidden,delta_Degree,degree);
       ForbiddenColors<<<n_blocks,n_threads>>>(V,preSum,colIndex,colors,d_vforbidden,CS);
       catchCudaError(hipMemcpy(checkConflict,checkConflict,sizeof(bool),hipMemcpyDeviceToHost),"conflict");
       
    }while(*checkConflict);
    gpu_time = clock()-gpu_time;

    //Assigned Colors
    /*
    @ last step to print the assigned colors
    */
    cout<<endl;
    for(long int i=0;i<V;i++){
       printf("vertex --> %i Assigned Color --> %d\n",i,colors[i]);
    }
    cout<<endl;
    printf("gpu_time is %.6lf\n",(double)gpu_time/((double)CLOCKS_PER_SEC/1000));
    
    thrust::device_ptr<long int> d_ptr = thrust::device_pointer_cast(colors);
    minimumColor = *(thrust::max_element(d_ptr, d_ptr+V));

    hipFree(checkConflict);
    hipFree(d_vforbidden);
    hipFree(CS);

    //required colors needed
    return minimumColor;
}

void GraphColoring_GPUAllocation(const char filename[]){
   
   //@difficult to allocate memory for large complete dataset not assume complete graph
   long int V; //No. of verties
   long int n_zero_counter = 0;   
   long int **st_Column;
   long int *st_degree;
   
   if(string(filename).find("col")!=string::npos){
     ReadColFile(filename,&V,&st_Column,&st_degree,&n_zero_counter);
   }else{
     ReadMMFile(filename,&V,&st_Column,&st_degree,&n_zero_counter); 
   }
   
   long int *degree;
   catchCudaError(hipMallocManaged(&degree,sizeof(long int)*V),"Degree Allocation");
   thrust::copy(st_degree,st_degree+V,degree);
   
   long int *d_preSum;
   catchCudaError(hipMallocManaged(&d_preSum,sizeof(long int)*(V+1)),"preSum Allocation");
   d_preSum[0] = 0;
   //store all the index of non zero element
   long int *d_colIndex;
   catchCudaError(hipMallocManaged(&d_colIndex,sizeof(long int)*n_zero_counter),"colIndex Allocation");
   
   //Allocation
   long int *colors;
   catchCudaError(hipMallocManaged(&colors,sizeof(long int)*V),"Color Allocation");
   long int *delta_Degree;
   catchCudaError(hipMallocManaged(&delta_Degree,sizeof(long int)),"Delta Degree Allocation");
   *delta_Degree = 0;
   
   preSumLength(V,d_preSum,degree,delta_Degree);

   for(int i=0;i<V;i++){
      //Remove the hipMemcpy it will take more time
      thrust::copy(st_Column[i],st_Column[i]+degree[i],d_colIndex+d_preSum[i]);
   }
   
   
   /*
   @begin CSR
   */
   long int width=16,height=16;
   long int threads_per_blocks = width*height;
   
   //Call the EdgeBase Algorithm
   long int number_Of_Colors_Needed = EdgeBased_Algorithm(V,d_preSum,d_colIndex,colors,degree,n_zero_counter,delta_Degree);  
   
   cout<<"EdgeBase Algorithm coloring found solution with "<<number_Of_Colors_Needed<<" colors"<<endl;
   cout<<"Valid coloring "; 
   
   bool *d_isValidColors; 
   catchCudaError(hipMallocManaged(&d_isValidColors,sizeof(bool)),"IsValid Allocation");
   *d_isValidColors = true;

   IsValidgraph_Coloring<<<ceil(V/threads_per_blocks)+1,threads_per_blocks>>>(V,colors,d_preSum,d_colIndex,d_isValidColors); 
   catchCudaError(hipMemcpy(d_isValidColors,d_isValidColors,sizeof(bool),hipMemcpyDeviceToHost),"IsValid Host Copy");
   
   if(*d_isValidColors){
     cout<<"yes"<<endl;
   }else{
     cout<<"No"<<endl;
   }
   
   //catchCudaError(hipDeviceSynchronize(),"GraphColoring DeviceSync");
   
   hipFree(d_preSum);
   hipFree(d_colIndex);
   hipFree(colors);
   hipFree(degree);
   hipFree(d_isValidColors);
   
}

/* Reading Argument with command line opetion */
int main(int argc,char *argv[])
{
     if(argc<2){
       cout<<"Invalid Input Parameter"<<endl;
       exit(1);
     }else{
      
     /*
     @Adding the clock
     */
     clock_t time = clock();
     GraphColoring_GPUAllocation(argv[1]);
     time = clock()-time; 
     
     cout<<"Total execution time is "<<(double)time/(double)CLOCKS_PER_SEC<<endl;
    
   }   

   return 0;
}
