#include "hip/hip_runtime.h"
/*
@Add sorting 3-opt
@Add heapSort
*/
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include<thrust/count.h>
#include<hiprand/hiprand_kernel.h>
#include<thrust/extrema.h>
#include<thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include "Utility.cuh"

#define MAXBLOCKS 1<<30

using namespace std;

//Heap Sort
__global__ void heapsortBasedOnRandom(long int V,long int *preSum,long int *colIndex,double *random){
  
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(threadId<V){

        long int n = preSum[threadId+1]-preSum[threadId];
        long int largest,previous,temp;
        
        for (int i = preSum[threadId]+n/2-1;i>=preSum[threadId];i--){ 
            
            largest = i,previous=-1;     
            while(largest!=previous){
              int l = 2*largest-preSum[threadId]+1;
              int r = 2*largest-preSum[threadId]+2;
              previous = largest;
              if(l<preSum[threadId+1]&&random[colIndex[l]]<random[colIndex[largest]]) 
                  largest = l; 
     
              if(r<preSum[threadId+1]&&random[colIndex[r]]<random[colIndex[largest]]) 
                largest = r; 
     
              if(largest!=previous) 
              { 
                temp = colIndex[previous];
                colIndex[previous] = colIndex[largest];
                colIndex[largest] = temp;
              }
           }
     
        }
 
        // One by one extract an element from heap 
        for (int i=preSum[threadId+1]-1;i>=preSum[threadId];i--) 
        { 
            temp = colIndex[preSum[threadId]];
            colIndex[preSum[threadId]] = colIndex[i];
            colIndex[i] = temp;
            largest = preSum[threadId],previous=-1;
            while(largest!=previous){
              int l = 2*largest-preSum[threadId]+1;
              int r = 2*largest-preSum[threadId]+2;
              previous = largest;
              if(l<i&&random[colIndex[l]]<random[colIndex[largest]]) 
                  largest = l; 
     
              if(r<i&&random[colIndex[r]]<random[colIndex[largest]]) 
                largest = r; 
     
              if(largest!=previous) 
              { 
                temp = colIndex[previous];
                colIndex[previous] = colIndex[largest];
                colIndex[largest] = temp;
             
              }
           }
     
        } 
    }

}

__global__ void generateRandom(hiprandState* globalState, double* randomArray,long int V,unsigned long seed) 
{
    long int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx<V){
       hiprandState localState = globalState[idx];
       hiprand_init(seed,idx,0,&localState);
       double RANDOM = hiprand_uniform(&localState);
       randomArray[idx] = RANDOM;
       globalState[idx] = localState;
    }
}

__global__ void assignColors(long int V,long int c,double *random,long int *colors,double *max_Array,
                             int *left){
   
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;

    if(threadId<V){
        if(colors[threadId]==-1){
            if(random[threadId]>max_Array[threadId]){
                colors[threadId] = c;
            }else{
                *left = 1;
            }
        }
    }
}

// How to arrange the program to assign the different color to adjacent
__global__ void graphColoringJPKernel(long int V,long int c,long int *preSum,long int *colIndex,double *random,long int *colors,double *max_Array){
     
     long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
     double max_Value = -1;
     if(threadId<V&&colors[threadId]!=-1){
        return;
     }
     else if(threadId<V){
           
         if(colors[threadId]==-1){
          
           max_Value = -1;

           for(long int k=preSum[threadId];k<preSum[threadId+1];k++){
             long int j = colIndex[k];
             long int jc = colors[j];
                 
             if(colors[j]!=-1){
               continue;
             }    

             if((threadId!=j&&colors[threadId]==colors[j])||(jc==-1)){        
               //check with random value
                if(random[j]>max_Value){
                    max_Value = random[j];
                    break;
                }
             }
             
           }

           max_Array[threadId] = max_Value;    
        }      
            
     }
} 

/*
@Calculation of PreSumLength
*/
void preSumLength(long int V,long int *d_preSum,long int *degree){
   
    for(long int i=0;i<V;i++){
       d_preSum[i+1] = d_preSum[i]+degree[i];
    }
}

__global__ void IsValidgraph_Coloring(long int V,long int *colors,long int *preSum,long int *colIndex,bool *flag){
   
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
   if(threadId<V){
       for(long int i=preSum[threadId];i<preSum[threadId+1];i++){
          if(colors[threadId]==colors[colIndex[i]]||colors[threadId]==-1){
             *flag = false;
          }
       }
   }
   
}

long int BaselineLabelingAlgorithm(long int V,long int *preSum,long int *colIndex,long int *colors){
    
    hiprandState* devStates;
    double *d_random;
    catchCudaError(hipMallocManaged(&d_random,sizeof(double)*V),"Random Allocation");
    double *d_maxArray;
    int *d_left,left;
    hipMallocManaged(&d_left,sizeof(int));
    //Allocate the memory for maxArray
    catchCudaError(hipMallocManaged(&d_maxArray,sizeof(double)*V),"maxArray Allocation");
    hipMallocManaged(&devStates,V*sizeof(hiprandState));

    //Step 1 assign the random value to all the vertices
    thrust::fill(d_maxArray,d_maxArray+V,-1);
    long int n_threads =  256;
    long int n_blocks = min((V+n_threads-1)/n_threads,(long)MAXBLOCKS);
    /*
    @ step 2 Initialize the colors to -1
    */
    thrust::fill(colors,colors+V,-1);
    long int cr = 1;
    
    clock_t gpu_time = clock();
    generateRandom<<<n_blocks,n_threads>>>(devStates,d_random,V,time(NULL));
    heapsortBasedOnRandom<<<n_blocks,n_threads>>>(V,preSum,colIndex,d_random);
    
    /*
    Call until all vetex are colored 
    */
    
    do{
       
       left = 0;     
       hipMemcpy(d_left,&left,sizeof(int),hipMemcpyHostToDevice);
       graphColoringJPKernel<<<n_blocks,n_threads>>>(V,cr,preSum,colIndex,d_random,colors,d_maxArray);
       assignColors<<<n_blocks,n_threads>>>(V,cr,d_random,colors,d_maxArray,d_left);             
       hipMemcpy(&left,d_left,sizeof(int),hipMemcpyDeviceToHost);
       
       cr++;

    }while(left);
    gpu_time = clock()-gpu_time;
    
    //Assigned Colors
    /*
    @ last step to print the assigned colors
    */
    printf("\n");
    for(long int i=0;i<V;i++){
       printf("vertex --> %i Assigned Color --> %d\n",i,colors[i]);
    }
    printf("\n");
    printf("gpu_time is %.6lf\n",(double)gpu_time/((double)CLOCKS_PER_SEC/1000));    
    
    //thrust::device_ptr<long int> d_ptr = thrust::device_pointer_cast(colors);
    //long int minimumColor = *(thrust::max_element(d_ptr, d_ptr+V));
    thrust::device_vector<long int> d_data(V);
    thrust::copy(colors,colors+V,d_data.begin());
    thrust::sort(d_data.begin(), d_data.end());

    size_t num_unique = thrust::inner_product(d_data.begin(), d_data.end()-1,d_data.begin()+1,0,
                                              thrust::plus<long int>(),thrust::not_equal_to<long int>())+1;
    
    hipFree(d_random);
    hipFree(d_maxArray);
    hipFree(devStates);
    hipFree(d_random);
    hipFree(d_left);
    
    return (long int)num_unique;
}

void GraphColoring_GPUAllocation(const char filename[]){
   
   //@Not Complete graph for large dataSet
   long int V; //No. of verties
   long int n_zero_counter = 0;
   long int **st_Column;
   long int *st_degree;   
  
   if(string(filename).find("col")!=string::npos||string(filename).find("clq")!=string::npos){
     ReadColFile(filename,&V,&st_Column,&st_degree,&n_zero_counter);
   }else{
     ReadMMFile(filename,&V,&st_Column,&st_degree,&n_zero_counter); 
   }
   
   long int *degree;
   catchCudaError(hipMallocManaged(&degree,sizeof(long int)*V),"Degree Allocation");
   thrust::copy(st_degree,st_degree+V,degree);
    
   long int *d_preSum;
   catchCudaError(hipMallocManaged(&d_preSum,sizeof(long int)*(V+1)),"preSum Allocation");
   d_preSum[0] = 0;
   //store all the index of non zero element
   long int *d_colIndex;
   catchCudaError(hipMallocManaged(&d_colIndex,sizeof(long int)*n_zero_counter),"colIndex Allocation");
   //Allocatio
   long int *colors;
   catchCudaError(hipMallocManaged(&colors,sizeof(long int)*V),"Color Allocation");

   //PreSum Length Calculation 
   preSumLength(V,d_preSum,degree);
   
   for(int i=0;i<V;i++){
      //Remove the hipMemcpy it will take more time
      thrust::copy(st_Column[i],st_Column[i]+degree[i],d_colIndex+d_preSum[i]);
   }   
   
   /*
   @begin CSR
   */
   long int width=16,height=16;
   long int threads_per_blocks = width*height;  
 
   //Call the Randomized Algorithm
   long int number_Of_Colors_Needed = BaselineLabelingAlgorithm(V,d_preSum,d_colIndex,colors);   

   printf("BaseLine Algorithm coloring found solution with %ld colors\n",number_Of_Colors_Needed);
   printf("Valid coloring ");
  
   bool *d_isValidColors; 
   catchCudaError(hipMallocManaged(&d_isValidColors,sizeof(bool)),"IsValid Allocation");
   *d_isValidColors = true;

   IsValidgraph_Coloring<<<ceil(V/(width*height))+1,threads_per_blocks>>>(V,colors,d_preSum,d_colIndex,d_isValidColors); 
   catchCudaError(hipMemcpy(d_isValidColors,d_isValidColors,sizeof(bool),hipMemcpyDeviceToHost),"Copy isValid Host");
   
   if(*d_isValidColors){
     printf("yes\n");
   }else{
     printf("No\n");
   }

   //catchCudaError(hipDeviceSynchronize(),"Graph Coloring Allocation");

   hipFree(d_preSum);
   hipFree(d_colIndex);
   hipFree(colors);
   hipFree(degree);
   //hipFree(d_isValidColors);

}

/* Reading Argument with command line opetion */
int main(int argc,char *argv[])
{
     if(argc<2){
       printf("Invalid Input Parameter\n");
       exit(1);
     }else{
      
     /*
     @Adding the clock
     */
     clock_t time = clock();
     GraphColoring_GPUAllocation(argv[1]);
     time = clock()-time; 
     
     double execution_time = (double)time/(double)CLOCKS_PER_SEC;
     printf("Total execution time is %lf\n",execution_time);
    
   }   

   return 0;
}
