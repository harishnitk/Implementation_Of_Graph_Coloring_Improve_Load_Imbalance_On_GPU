#include "hip/hip_runtime.h"
/*
@Optimization of Hybrid-6-opt, remove variables
Need to optimize
@Add Sorting,heapSort
*/
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include<thrust/count.h>
#include<hiprand/hiprand_kernel.h>
#include<thrust/extrema.h>
#include<thrust/device_ptr.h>
#include "Utility.cuh"

#define MAXBLOCKS 1<<30

using namespace std;


//Heap Sort
__global__ void heapsortBasedOnDegree(long int V,long int *preSum,long int *colIndex,long int *degree){
  
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(threadId<V){

        long int n = preSum[threadId+1]-preSum[threadId];
        long int largest,previous,temp;
        
        for (int i = preSum[threadId]+n/2-1;i>=preSum[threadId];i--){ 
            
            largest = i,previous=-1;     
            while(largest!=previous){
              int l = 2*largest-preSum[threadId]+1;
              int r = 2*largest-preSum[threadId]+2;
              previous = largest;
              if(l<preSum[threadId+1]&&degree[colIndex[l]]<degree[colIndex[largest]]) 
                  largest = l; 
     
              if(r<preSum[threadId+1]&&degree[colIndex[r]]<degree[colIndex[largest]]) 
                largest = r; 
     
              if(largest!=previous) 
              { 
                temp = colIndex[previous];
                colIndex[previous] = colIndex[largest];
                colIndex[largest] = temp;
              }
           }
     
        }
 
        // One by one extract an element from heap 
        for (int i=preSum[threadId+1]-1;i>=preSum[threadId];i--) 
        { 
            temp = colIndex[preSum[threadId]];
            colIndex[preSum[threadId]] = colIndex[i];
            colIndex[i] = temp;
            largest = preSum[threadId],previous=-1;
            while(largest!=previous){
              int l = 2*largest-preSum[threadId]+1;
              int r = 2*largest-preSum[threadId]+2;
              previous = largest;
              if(l<i&&degree[colIndex[l]]<degree[colIndex[largest]]) 
                  largest = l; 
     
              if(r<i&&degree[colIndex[r]]<degree[colIndex[largest]]) 
                largest = r; 
     
              if(largest!=previous) 
              { 
                temp = colIndex[previous];
                colIndex[previous] = colIndex[largest];
                colIndex[largest] = temp;
             
              }
           }
     
        } 
    }

}

//Heap Sort
__global__ void heapsortBasedOnRandom(long int V,long int *preSum,long int *colIndex,double *random){
  
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(threadId<V){

        long int n = preSum[threadId+1]-preSum[threadId];
        long int largest,previous,temp;
        
        for (int i = preSum[threadId]+n/2-1;i>=preSum[threadId];i--){ 
            
            largest = i,previous=-1;     
            while(largest!=previous){
              int l = 2*largest-preSum[threadId]+1;
              int r = 2*largest-preSum[threadId]+2;
              previous = largest;
              if(l<preSum[threadId+1]&&random[colIndex[l]]<random[colIndex[largest]]) 
                  largest = l; 
     
              if(r<preSum[threadId+1]&&random[colIndex[r]]<random[colIndex[largest]]) 
                largest = r; 
     
              if(largest!=previous) 
              { 
                temp = colIndex[previous];
                colIndex[previous] = colIndex[largest];
                colIndex[largest] = temp;
              }
           }
     
        }
 
        // One by one extract an element from heap 
        for (int i=preSum[threadId+1]-1;i>=preSum[threadId];i--) 
        { 
            temp = colIndex[preSum[threadId]];
            colIndex[preSum[threadId]] = colIndex[i];
            colIndex[i] = temp;
            largest = preSum[threadId],previous=-1;
            while(largest!=previous){
              int l = 2*largest-preSum[threadId]+1;
              int r = 2*largest-preSum[threadId]+2;
              previous = largest;
              if(l<i&&random[colIndex[l]]<random[colIndex[largest]]) 
                  largest = l; 
     
              if(r<i&&random[colIndex[r]]<random[colIndex[largest]]) 
                largest = r; 
     
              if(largest!=previous) 
              { 
                temp = colIndex[previous];
                colIndex[previous] = colIndex[largest];
                colIndex[largest] = temp;
             
              }
           }
     
        } 
    }

}


__global__ void generateRandom(hiprandState* globalState, double* randomArray,long int V,unsigned long seed) 
{
    long int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx<V){
       hiprandState localState = globalState[idx];
       hiprand_init(seed,idx,0,&localState);
       double RANDOM = hiprand_uniform(&localState);
       randomArray[idx] = RANDOM;
       globalState[idx] = localState;
    }
}


__global__ void assignColors(long int V,long int c,double *random,long int *colors,double *max_Array,
                             int *left){
   
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;

    if(threadId<V){
        if(colors[threadId]==-1){
            if(random[threadId]>max_Array[threadId]){
                colors[threadId] = c;
            }else{
                *left = 1;
            }
        }
    }
}

__global__ void assign_DegreeColors(long int V,long int c,long int *degree,bool *flag,long int *colors,
                                    long int *max_Degree,bool *isDegree,int *left){
    
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(threadId<V){
        if(colors[threadId]==-1){
           if(flag[threadId]==false){
              if(degree[threadId]>max_Degree[threadId]){
                  colors[threadId] = c;
                  *isDegree = true;
                  //reAssign Values
              }else{
                *left = 1;
              } 
           }else{
              flag[threadId] = false;
              *left = 1;
           }
        }
    }
}

//Degree Coloring
__global__ void degree_Coloring(long int V,long int c,long int *preSum,long int *colIndex,
                                long int *degree,long int *colors,long int *max_Degree,bool *d_flag){
  
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
   long int maxDegree = -1;

   if(threadId<V&&colors[threadId]!=-1){
      return;
   }
   else if(threadId<V){
            
       if(colors[threadId]==-1){
          
           maxDegree = -1;
          
           for(long int k=preSum[threadId];k<preSum[threadId+1];k++){
               long int j=colIndex[k];
               long int jc = colors[j];
               
               if(colors[j]!=-1){
                 continue;
               }
              
              //check all neighborhood which all are uncolored
              if(jc==-1){
                 if(degree[threadId]==degree[j]){
                    d_flag[threadId] = true;
                    break;  
                 }else if(degree[j]>maxDegree){
                   maxDegree = degree[j];
                   break;
                 }
              }
           }              
           
           max_Degree[threadId] = maxDegree;
       }
        
        
   }

}

// How to arrange the program to assign the different color to adjacent
__global__ void randomized_Based_Label(long int V,long int c,long int *preSum,long int *colIndex,
                                       double *random,long int *colors,double *max_Array){
     
     long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
     double max_Value = -1;
     
     if(threadId<V&&colors[threadId]!=-1){
        return;
     }
     else if(threadId<V){
        
         if(colors[threadId]==-1){
          
           max_Value = -1;

           for(long int k=preSum[threadId];k<preSum[threadId+1];k++){
             long int j = colIndex[k];
             long int jc = colors[j];
             
             if(colors[j]!=-1){
               continue;
             }

             if((threadId!=j&&colors[threadId]==colors[j])||(jc==-1)){        
               //check with random value
                if(random[j]>max_Value){
                    max_Value = random[j];
                    break;
                }
             }
             
           }
       
           max_Array[threadId] = max_Value;

          }     
         
     }
} 

void preSumLength(int V,long int *d_preSum,long int *degree){
   
    for(long int i=0;i<V;i++){
       d_preSum[i+1] = d_preSum[i]+degree[i];
    }

}

__global__ void IsValidgraph_Coloring(long int V,long int *colors,long int *preSum,long int *colIndex,bool *flag){
   
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
   if(threadId<V){
       for(long int i=preSum[threadId];i<preSum[threadId+1];i++){
          if(colors[threadId]==colors[colIndex[i]]||colors[threadId]==-1){
             *flag = false;
          }
       }
   }
   
}


long int Hybrid_Algorithm(long int V,long int *preSum,long int *colIndex,long int *colors,long int *degree){
    
    hiprandState* devStates;
    double *d_random;
    hipMallocManaged(&d_random,sizeof(double)*V);
    double *d_maxArray;
    long int *d_maxDegree;
    bool *d_flag;
    bool *d_hybrid,hybrid;
    
    //Allocate the memory for maxArray
    catchCudaError(hipMallocManaged(&d_maxArray,sizeof(double)*V),"MaxArray Allocation");
    catchCudaError(hipMallocManaged(&d_maxDegree,sizeof(long int)*V),"MaxDegree Allocation");
    catchCudaError(hipMallocManaged(&d_flag,sizeof(bool)*V),"Flag Allocation");
    catchCudaError(hipMallocManaged(&d_hybrid,sizeof(bool)),"Hybrid Allocation");
    hipMallocManaged(&devStates,V*sizeof(hiprandState));
    
    int *d_left,left;
    hipMallocManaged(&d_left,sizeof(int));
    
    long int n_threads =  256;
    long int n_blocks = min((V+n_threads-1)/n_threads,(long)MAXBLOCKS);
    //Default case;
    hybrid = true;
    
     /*
    @ step 2 Initialize the colors to -1
    @ until all are colored
    */
    thrust::fill(colors,colors+V,-1);
    long int cr = 1;

    generateRandom<<<n_blocks,n_threads>>>(devStates,d_random,V,time(NULL));
    heapsortBasedOnDegree<<<n_blocks,n_threads>>>(V,preSum,colIndex,degree);

    /*
      Call until all vetex are colored 
    */ 
    clock_t gpu_time = clock();
    do{      
        
       //For Degree Coloring
       if(hybrid){
          left = 0;
          hybrid = false;
          catchCudaError(hipMemcpy(d_hybrid,&hybrid,sizeof(bool),hipMemcpyHostToDevice),"ishybrid Copy");
          hipMemcpy(d_left,&left,sizeof(int),hipMemcpyHostToDevice);
          degree_Coloring<<<n_blocks,n_threads>>>(V,cr,preSum,colIndex,degree,colors,d_maxDegree,d_flag);       
          assign_DegreeColors<<<n_blocks,n_threads>>>(V,cr,degree,d_flag,colors,d_maxDegree,d_hybrid,d_left);
          catchCudaError(hipMemcpy(&hybrid,d_hybrid,sizeof(bool),hipMemcpyDeviceToHost),"ishybrid Copy");
          hipMemcpy(&left,d_left,sizeof(int),hipMemcpyDeviceToHost);
           
          if(hybrid==true){
            cr++;
          }else{
            heapsortBasedOnRandom<<<n_blocks,n_threads>>>(V,preSum,colIndex,d_random);
          }   
       }else{
      
         //For Randomized Based Labelling
         left = 0;
         hipMemcpy(d_left,&left,sizeof(int),hipMemcpyHostToDevice);
         randomized_Based_Label<<<n_blocks,n_threads>>>(V,cr,preSum,colIndex,d_random,colors,d_maxArray);       
         assignColors<<<n_blocks,n_threads>>>(V,cr,d_random,colors,d_maxArray,d_left);
         hipMemcpy(&left,d_left,sizeof(int),hipMemcpyDeviceToHost);
         cr++;
       }             

    
    }while(left);
    gpu_time = clock()-gpu_time;
    //Assigned Colors
    /*
    @ last step to print the assigned colors
    */
    printf("\n");
    for(long int i=0;i<V;i++){
       printf("vertex --> %i Assigned Color --> %d\n",i,colors[i]);
    }
    printf("\n");
    printf("gpu_time is %.6lf\n",(double)gpu_time/((double)CLOCKS_PER_SEC/1000));       

    //thrust::device_ptr<long int> d_ptr = thrust::device_pointer_cast(colors);
    //long int minimumColor = *(thrust::max_element(d_ptr, d_ptr+V));
    thrust::device_vector<long int> d_data(V);
    thrust::copy(colors,colors+V,d_data.begin());
    thrust::sort(d_data.begin(), d_data.end());

    size_t num_unique = thrust::inner_product(d_data.begin(), d_data.end()-1,d_data.begin()+1,0,
                                              thrust::plus<long int>(),thrust::not_equal_to<long int>())+1;

    hipFree(d_random);
    hipFree(d_flag);
    hipFree(d_maxArray);
    hipFree(d_maxDegree);
    hipFree(d_hybrid);
    hipFree(d_left);

    return (long int)num_unique;
}

void GraphColoring_GPUAllocation(const char filename[]){
   
   //@difficult to allocate memory for large complete dataset not assume complete graph
   long int V; //No. of verties
   long int n_zero_counter = 0;   
   long int **st_Column;
   long int *st_degree;
   
   if(string(filename).find("col")!=string::npos||string(filename).find("clq")!=string::npos){
     ReadColFile(filename,&V,&st_Column,&st_degree,&n_zero_counter);
   }else{
     ReadMMFile(filename,&V,&st_Column,&st_degree,&n_zero_counter); 
   }
   
   long int *degree;
   catchCudaError(hipMallocManaged(&degree,sizeof(long int)*V),"Degree Allocation");
   thrust::copy(st_degree,st_degree+V,degree);
   
   long int *d_preSum;
   catchCudaError(hipMallocManaged(&d_preSum,sizeof(long int)*(V+1)),"preSum Allocation");
   d_preSum[0] = 0;
   //store all the index of non zero element
   long int *d_colIndex;
   catchCudaError(hipMallocManaged(&d_colIndex,sizeof(long int)*n_zero_counter),"colIndex Allocation");
   //Allocatio
   long int *colors;
   catchCudaError(hipMallocManaged(&colors,sizeof(long int)*V),"Color Allocation");

   preSumLength(V,d_preSum,degree);

   for(int i=0;i<V;i++){
      //Remove the hipMemcpy it will take more time
      thrust::copy(st_Column[i],st_Column[i]+degree[i],d_colIndex+d_preSum[i]);
   }  
   
   /*
   @begin CSR
   */
   long int width=16,height=16;
   long int threads_per_blocks = width*height;
   
   //Call the Hybrid Algorithm
   long int number_Of_Colors_Needed = Hybrid_Algorithm(V,d_preSum,d_colIndex,colors,degree);  
   
   printf("Hybrid Algorithm coloring found solution with %ld colors\n",number_Of_Colors_Needed);
   printf("Valid coloring "); 
   
   bool *d_isValidColors; 
   catchCudaError(hipMallocManaged(&d_isValidColors,sizeof(bool)),"IsValid Allocation");
   *d_isValidColors = true;

   IsValidgraph_Coloring<<<ceil(V/(width*height))+1,threads_per_blocks>>>(V,colors,d_preSum,d_colIndex,d_isValidColors); 
   catchCudaError(hipMemcpy(d_isValidColors,d_isValidColors,sizeof(bool),hipMemcpyDeviceToHost),"Copy isValid Host");
   
   if(*d_isValidColors){
     printf("yes\n");
   }else{
     printf("No\n");
   }
   
   //catchCudaError(hipDeviceSynchronize(),"GraphColoring DeviceSync");

   hipFree(d_preSum);
   hipFree(d_colIndex);
   hipFree(colors);
   hipFree(degree);
   hipFree(d_isValidColors);
   
}

/* Reading Argument with command line opetion */
int main(int argc,char *argv[])
{
     if(argc<2){
       printf("Invalid Input Parameter\n");;
       exit(1);
     }else{
      
     /*
     @Adding the clock
     */
     clock_t time = clock();
     GraphColoring_GPUAllocation(argv[1]);
     time = clock()-time;
     double execution_time = (double)time/(double)CLOCKS_PER_SEC;
     
     printf("Total execution time is %lf\n",execution_time);
    
   }   

   return 0;
}
