#include "hip/hip_runtime.h"
/*
Need to add virtual wrap centric programming
modified on 4-may-2019, try random bit selection
*/
#include<bits/stdc++.h>
#include<hip/hip_runtime.h>
#include<thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include<hiprand/hiprand_kernel.h>
#include<thrust/extrema.h>
#include<thrust/device_ptr.h>
#include "Utility.cuh"

#define MAXBLOCKS 1<<30
#define MAXDELTA 1000000000

using namespace std;

/*
[method] {find the unsetRandomColor}
[]clear the neighbour bit
*/
__device__ long int findUnsetRandomColor(long int Id,unsigned int *Colorset,long int maxClr){
    
    long int j,phase,end=(maxClr/32)+1,bit,left=maxClr%32;
    
    for(phase=0;phase<end;phase++){
      
      if(Id%2==0){
        bit = __ffs(Colorset[Id*end+phase]);
      }else{
         if(phase==end-1&&left!=0){
           unsigned int Value = 0;
           j = left-1;
           while(Colorset[Id*end+phase]>0){
             Value = Value+(Colorset[Id*end+phase])%2*(1<<j);
             j--;
             Colorset[Id*end+phase] = Colorset[Id*end+phase]/2;
           }
           bit = __ffs(Value);
           bit = bit!=0?left+1-bit:bit;
         }else{
            Colorset[Id*end+phase] = __brev(Colorset[Id*end+phase]);
            bit = __ffs(Colorset[Id*end+phase]);
            bit = bit!=0?33-bit:bit;
         }
      }
      
      if(bit!=0){
        return phase*32+bit;
      }

    }
    
    return -1;

}

__global__ void findtheNeighbourColors(long int V,long int *colors,long int *preSum,long int 
                                       *colIndex,unsigned int *Colorset,long int maxClr){
    
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;

    if(threadId<V){

        long int k=preSum[threadId],n=preSum[threadId+1],j,phase,end=(maxClr/32)+1,left=maxClr%32;
        Colorset[threadId*end+end-1] = left!=0?(1<<left)-1:4294967295;//32 bit number
        
        for(phase=0;phase<end-1;phase++){
          Colorset[threadId*end+phase] = 4294967295;
        }
        
        for(;k<n;k++,n--){
           j = colIndex[k];
           
           if(colors[j]!=0){
              phase = colors[j]/32;
              if(colors[j]%32==0){
                phase--;
              }
              Colorset[threadId*end+phase]&= ~(unsigned int)(1<<(colors[j]-1-phase*32));//clear bit
           }

           j = colIndex[n-1];
           if(colors[j]!=0){
              phase = colors[j]/32;
              if(colors[j]%32==0){
                phase--;
              }
              Colorset[threadId*end+phase]&= ~(unsigned int)(1<<(colors[j]-1-phase*32));//clear bit
           }

        }
    }

}

/*
[method] {AssignColor}
{Description} []
*/
__global__ void assignColors(long int V,long int *colors,long int *colIndex,long int *preSum,unsigned int 
                             *Colorset,long int *degree,bool *inc,long int maxClr){
   
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;

    if(threadId<V&&colors[threadId]==0){
           
       //Select first available colors Intimation of available color which is minimum
       if(degree[threadId]==0){
         colors[threadId] = 1; //{to reduce the colors}
       }else{
          //choose color form from its degree
          colors[threadId] = findUnsetRandomColor(threadId,Colorset,maxClr); 

          if(colors[threadId]==-1){
             *inc = true;
             colors[threadId] = 0;
          }
       }        

    }

}

/*
@reset colors to 0
*/
__global__ void DetectConflictsColors(long int V,long int *preSum,long int *colIndex,long int *colors,
                                      long int *degree,bool *checkConflict,int isDegreeChange){
  
    long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(threadId<V){

      if(colors[threadId]==0){
        *checkConflict = true;
        return;
      }
    
      for(long int k=preSum[threadId];k<preSum[threadId+1]&&colors[threadId]!=0;k++){
         
         long int j = colIndex[k];
         
         if(colors[j]==0){
            continue;
         }

         if(isDegreeChange){
            if((colors[threadId]==colors[j])&&(degree[threadId]>degree[j])){
              colors[j] = 0;
              *checkConflict = true;
              break;
            }else if((colors[threadId]==colors[j])&&(degree[threadId]<degree[j])){
              colors[threadId] = 0;
              *checkConflict = true;
              break;
            }else if((colors[threadId]==colors[j])&&(threadId>j)){
              colors[j] = 0;
              *checkConflict = true;
              break;
            }else if((colors[threadId]==colors[j])&&(threadId<j)){
              colors[threadId] = 0;
              *checkConflict = true;
              break;
            }
         }else if((colors[threadId]==colors[j])&&(threadId>j)){
              colors[j] = 0;
              *checkConflict = true;
              break;
         }else if((colors[threadId]==colors[j])&&(threadId<j)){
              colors[threadId] = 0;
              *checkConflict = true;
              break;
         }

      }

    }

}

__global__ void IsValidgraph_Coloring(long int V,long int *colors,long int *preSum,long int *colIndex,
                                      bool *flag){
   
   long int threadId = blockDim.x*blockIdx.x+threadIdx.x;
   if(threadId<V){
       for(long int i=preSum[threadId];i<preSum[threadId+1];i++){
          if(colors[threadId]==colors[colIndex[i]]||colors[threadId]<=0){
             *flag = false;
          }
       }
   }
   
}

void preSumLength(int V,long int *d_preSum,long int *degree,long int &deltaDegree,long int &isDegreeChange){
    
    for(long int i=0;i<V;i++){
       d_preSum[i+1] = d_preSum[i]+degree[i];
       if(deltaDegree<degree[i]){
          deltaDegree = degree[i];
       }

       if((i+1)<V&&(degree[i]!=degree[i+1])){
         isDegreeChange = 1;
       }
    }

    deltaDegree+= 1;

}

long int ConflictColorAlgorithm(long int V,long int *preSum,long int *colIndex,long int *colors,
                                long int *degree,int deltaDegree,int isDegreeChange){
    
    double *d_random;
    hipMallocManaged(&d_random,sizeof(double)*V);
    unsigned int *d_Colorset;
    bool *d_checkConflict,checkConflict,*d_inc,inc;

    //Allocate the memory for maxArray
    catchCudaError(hipMallocManaged(&d_checkConflict,sizeof(int)),"checkConflict Allocation");
    hipMallocManaged(&d_Colorset,sizeof(unsigned int)*(min((long)MAXDELTA,(long)V*(deltaDegree/32+1))));
    catchCudaError(hipMallocManaged(&d_checkConflict,sizeof(bool)),"cc");
    catchCudaError(hipMallocManaged(&d_inc,sizeof(bool)),"inc");

    long int n_threads =  256;
    long int n_blocks = min((V+n_threads-1)/n_threads,(long)MAXBLOCKS);
    long int maxClr = 2;
    
    /*
    @ step 2 Initialize the colors to 0
    @ until all are colored
    */
    thrust::fill(colors,colors+V,0);
    
    clock_t gpu_time = clock();
    do{
         
         checkConflict = false;
         inc = false;
         catchCudaError(hipMemcpy(d_checkConflict,&checkConflict,sizeof(bool),hipMemcpyHostToDevice),"cc");
         catchCudaError(hipMemcpy(d_inc,&inc,sizeof(bool),hipMemcpyHostToDevice),"inc copy");
         
         findtheNeighbourColors<<<n_blocks,n_threads>>>(V,colors,preSum,colIndex,d_Colorset,maxClr);
         assignColors<<<n_blocks,n_threads>>>(V,colors,colIndex,preSum,d_Colorset,degree,d_inc,maxClr);
         DetectConflictsColors<<<n_blocks,n_threads>>>(V,preSum,colIndex,colors,degree,d_checkConflict,isDegreeChange);       
         
         catchCudaError(hipMemcpy(&inc,d_inc,sizeof(bool),hipMemcpyDeviceToHost),"inc copy");
         catchCudaError(hipMemcpy(&checkConflict,d_checkConflict,sizeof(bool),hipMemcpyDeviceToHost),"cc");
         
         if(inc){
            maxClr = 2*maxClr;
         }

    
    }while(checkConflict);
    gpu_time = clock()-gpu_time;
    //Assigned Colors
    /*
    @ last step to print the assigned colors
    */
    printf("\n");
    for(long int i=0;i<V;i++){
       printf("vertex --> %i Assigned Color --> %d\n",i,colors[i]);
    }
    printf("\n");
    printf("gpu_time is %.6lf\n",(double)gpu_time/((double)CLOCKS_PER_SEC/1000));

    thrust::device_vector<long int> d_data(V);
    thrust::copy(colors,colors+V,d_data.begin());
    thrust::sort(d_data.begin(), d_data.end());

    size_t num_unique = thrust::inner_product(d_data.begin(), d_data.end()-1,d_data.begin()+1,0,
                                              thrust::plus<long int>(),thrust::not_equal_to<long int>())+1;
    

    hipFree(d_random);
    hipFree(d_checkConflict);
    hipFree(d_inc);
    hipFree(d_Colorset);

    //required colors needed
    return (long int)num_unique;
}

void GraphColoring_GPUAllocation(const char filename[]){
   
   //@difficult to allocate memory for large complete dataset not assume complete graph
   long int V,deltaDegree = 0,isDegreeChange = 0; //No. of verties
   long int n_zero_counter = 0;   
   long int **st_Column;
   long int *st_degree;
   
   if(string(filename).find("col")!=string::npos||string(filename).find("clq")!=string::npos){
     ReadColFile(filename,&V,&st_Column,&st_degree,&n_zero_counter);
   }else{
     ReadMMFile(filename,&V,&st_Column,&st_degree,&n_zero_counter); 
   }
 
   long int *degree;
   catchCudaError(hipMallocManaged(&degree,sizeof(long int)*V),"Degree Allocation");
   thrust::copy(st_degree,st_degree+V,degree);
   
   long int *d_preSum;
   catchCudaError(hipMallocManaged(&d_preSum,sizeof(long int)*(V+1)),"preSum Allocation");
   d_preSum[0] = 0;
   //store all the index of non zero element
   long int *d_colIndex;
   catchCudaError(hipMallocManaged(&d_colIndex,sizeof(long int)*n_zero_counter),"colIndex Allocation");
   
   //Allocation
   long int *colors;
   catchCudaError(hipMallocManaged(&colors,sizeof(long int)*V),"Color Allocation");

   preSumLength(V,d_preSum,degree,deltaDegree,isDegreeChange);

   for(int i=0;i<V;i++){
      //Remove the hipMemcpy it will take more time
      thrust::copy(st_Column[i],st_Column[i]+degree[i],d_colIndex+d_preSum[i]);
   }  
   
   /*
   @begin CSR
   */
   
   //Call the Color Conflict Algorithm
   long int number_Of_Colors_Needed = ConflictColorAlgorithm(V,d_preSum,d_colIndex,colors,degree,deltaDegree,isDegreeChange);  
   
   printf("ConflictColor Algorithm coloring found solution with %d colors\n",number_Of_Colors_Needed);
   printf("Valid coloring Yes\n"); 
   
   //catchCudaError(hipDeviceSynchronize(),"GraphColoring DeviceSync");
   
   hipFree(d_preSum);
   hipFree(d_colIndex);
   hipFree(colors);
   hipFree(degree);
  
}

/* Reading Argument with command line opetion */
int main(int argc,char *argv[])
{
     if(argc<2){
       printf("Invalid Input Parameter\n");
       exit(1);
     }else{

     /*
     @Adding the clock
     */
     clock_t time = clock();
     GraphColoring_GPUAllocation(argv[1]);
     time = clock()-time; 
     
     double execution_time = (double)time/(double)CLOCKS_PER_SEC;
     
     printf("Total execution time is %lf\n",execution_time);
    
   }   

   return 0;
}
