#include<bits/stdc++.h>
#include "hip/hip_runtime.h"
using namespace std;

void catchCudaError(hipError_t error, const char *function)
{
    if(error!=hipSuccess)
    {
        printf("\n====== Cuda Error Code %i ======\n %s in CUDA %s\n", error, hipGetErrorString(error), function);
        exit(-1);
    }
}

void ReadColFile(const char filename[],long int *V,long int ***st_Column,long int **st_degree,long int *counter){
   string s;
   ifstream infile(filename);
   if(infile.fail()){
      cout<<"Fail to open the file\n";
      exit(0);
   }

   long int n_rows,n_edges;
   //Maintain Hash for NNZ,preSum,colIndex
   //allocate dynamic size

   while(getline(infile,s)){
       istringstream iss(s);
       string str;
       long int u,v;
       iss>>str;
       if(str=="p"){
          iss>>s;
          iss>>n_rows;
          iss>>n_edges;
          *V = n_rows;
          *st_degree = new long int[n_rows];
          *st_Column = new long int*[n_rows];
          memset(*st_degree,0,n_rows*sizeof(long int));
          continue;
       }else if(str!="e"){
          continue; 
       }

       iss>>u>>v;
         if(u!=v){
         long int u_len = (*st_degree)[u-1];
         long int v_len = (*st_degree)[v-1];
         (*st_Column)[u-1] = (long int*)realloc((*st_Column)[u-1],sizeof(long int)*(u_len+1));
         (*st_Column)[v-1] = (long int*)realloc((*st_Column)[v-1],sizeof(long int)*(v_len+1));
         (*st_Column)[u-1][u_len] = v-1; 
         (*st_Column)[v-1][v_len] = u-1;
         (*st_degree)[u-1]++;
         (*st_degree)[v-1]++;
         *counter+=2;
       }
        
   }

   infile.close();
}

void ReadMMFile(const char filename[], long int *V,long int ***st_Column,long int **st_degree,long int *counter){
   string s;
   ifstream infile(filename);
   if(infile.fail()){
      cout<<"Fail to open the file\n";
      return;
   }

   //content
   while(getline(infile,s)){
     istringstream iss(s);
     if(s.find("%")==string::npos){
        break;
     }
   }

   istringstream iss(s);
   //Maintain Hash for NNZ,preSum,colIndex
   //allocate dynamic size

   long int n_rows,n_cols,n_edges;
   iss>>n_rows>>n_cols>>n_edges;
   *st_degree = new long int[n_rows];
   *st_Column = new long int*[n_rows];
   memset(*st_degree,0,n_rows*sizeof(long int));
   *V = n_rows;

   //reading edges

   while(getline(infile,s)){
      istringstream iss(s);
      long int u,v,w;
      iss>>u>>v>>w;
      if(u!=v){
        long int u_len = (*st_degree)[u-1];
        long int v_len = (*st_degree)[v-1];
        (*st_Column)[u-1] = (long int*)realloc((*st_Column)[u-1],sizeof(long int)*(u_len+1));
        (*st_Column)[v-1] = (long int*)realloc((*st_Column)[v-1],sizeof(long int)*(v_len+1));
        (*st_Column)[u-1][u_len] = v-1; 
        (*st_Column)[v-1][v_len] = u-1;
        (*st_degree)[u-1]++;
        (*st_degree)[v-1]++;
        *counter+=2;
      }
   }

   infile.close();
}
